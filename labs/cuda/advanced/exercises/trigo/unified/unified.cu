#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../utils.h"

#define NUM_ELEM (16 * 1024 * 1024)

__global__ void kernel_compute(float* a, float* b, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_UnifiedMem(void) {
  float* host_a = 0;
  float* host_b = 0;

  // TODO-unified: Alocare memorie unificata
  hipMallocManaged(&host_a, NUM_ELEM * sizeof(float));
  hipMallocManaged(&host_b, NUM_ELEM * sizeof(float));

  if (host_a == 0 || host_b == 0) {
    printf("[HOST] Couldn't allocate memory\n");
    return 1;
  }

  fill_array_random(host_b, NUM_ELEM);

  kernel_compute<<<NUM_ELEM / 256, 256>>>(host_a, host_b, NUM_ELEM);
  hipDeviceSynchronize();

  // TODO print first partial results
  print_array_partial(host_a, NUM_ELEM);

  // TODO-unified: Dealocare memorie unificata
  hipFree(host_a);
  hipFree(host_b);

  return 0;
}

int main(void) { compute_UnifiedMem(); }
